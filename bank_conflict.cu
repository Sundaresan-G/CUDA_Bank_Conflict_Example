
#include <hip/hip_runtime.h>
#include <iostream>

#define N 2

#define number int

using namespace std;

__global__ void add(number *a){
    __shared__ number sharedA[N + 2048];
    // __shared__ number sharedB[N + 2048];

    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;   

    int stride = 0;

    if (index != 0)
        stride = index*32 - index%32;

    // if (index %2 == 0)
    //     stride = 33;
    // else 
    //     stride = 0;
    sharedA[index + stride] = index;

    // for (stride = 1; stride < 66; stride++)  
    // sharedA[index+stride] = index+stride;  
    // sharedA[index + 1] = sharedA[index + 1 + stride];

    // if (index + stride < N)
    //     sharedA[index] = sharedA[index] + sharedA[index + stride];

    __syncthreads();

    if (index == 0){
        // if (sharedB[0] == 'c')
        *a = sharedA[index];
    }
    
}

int main(){

    number *da, a=0;

    hipMalloc((void **)&da, sizeof(number));

    int num_threads = N;

    add<<<1, num_threads>>>(da);  

    hipMemcpy(&a, da, sizeof(number), hipMemcpyDeviceToHost);

    cout << a << endl;

}
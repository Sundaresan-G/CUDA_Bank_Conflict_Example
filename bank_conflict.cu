
#include <hip/hip_runtime.h>
#include <iostream>

#define number float

constexpr int THREADS_PER_BLOCK=256;

constexpr size_t N = (size_t)8192*8192;

// Issue 1 to avoid all bank conflicts or 32 in case of int or float to get max bank conflicts
int OFFSET = 1;

using namespace std;

__global__ void bankConflictKernel(number *a, int OFFSET){
    extern __shared__ number sharedA[];
    // __shared__ number sharedB[N + 2048];

    size_t index = blockDim.x * blockIdx.x + threadIdx.x;   

    if (index >= N) return;

    int stride = threadIdx.x*OFFSET;

    // if (index != 0)
    //     stride = index*32 - index%32;

    // if (index %2 == 0)
    //     stride = 33;
    // else 
    //     stride = 0;
    sharedA[stride] = index;

    // for (stride = 1; stride < 66; stride++)  
    // sharedA[index+stride] = index+stride;  
    // sharedA[index + 1] = sharedA[index + 1 + stride];

    // if (index + stride < N)
    //     sharedA[index] = sharedA[index] + sharedA[index + stride];

    __syncthreads();

    // This is included so that compiler does not eliminate the dead code
    // if (index == 0){
    //     // if (sharedB[0] == 'c')
    //     *a = sharedA[index];
    // }

    a[index] = sharedA[stride];
    
}

int main(){

    number *da;

    hipMalloc((void **)&da, sizeof(number) * N);

    size_t num_threads = THREADS_PER_BLOCK;

    size_t num_blocks = (N + num_threads - 1)/num_threads;

    // Warmup
    // 8 * sizeof(number) is 32 or 64 depending on float or double
    bankConflictKernel<<<num_blocks, num_threads, THREADS_PER_BLOCK * 8 * sizeof(number) * sizeof(number)>>>(da, OFFSET); 
    // cudaDeviceSynchronize();

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // cudaDeviceSynchronize();

    cout << "Total Elapsed time for " << N << " elements is \n";
    printf("Offset ElapsedTime\n");

    for (OFFSET = 1; OFFSET <= 32; OFFSET ++){

        hipEventRecord(start, 0);
        bankConflictKernel<<<num_blocks, num_threads, THREADS_PER_BLOCK * 8 * sizeof(number) * sizeof(number)>>>(da, OFFSET);  
        hipEventRecord(stop, 0);

        hipEventSynchronize(stop);

        float elapsedTime = 0;

        hipEventElapsedTime(&elapsedTime, start, stop);

        printf("%6d %10.8f\n", OFFSET, elapsedTime);

        // cout << "Total Elapsed time for " << N << " elements with offset " << OFFSET << " is "<<elapsedTime <<" ms\n";
    }

    return 0;

}